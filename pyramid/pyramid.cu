#include "hip/hip_runtime.h"
/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
% by: Alireza Ahmadi                                     %
% University of Bonn- MSc Robotics & Geodetic Engineering%
% Alireza.Ahmadi@uni-bonn.de                             %
% AlirezaAhmadi.xyz                                      %
%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/
#include "pyramid.h"

namespace DynaMap{

    pyramid::~pyramid(){}

    // __global__
    // void upSampleKernel(float *src, float *dst){
    // }
    __global__
    void downSampleKernel(float *src, float *dst, rgbdSensor sensor,int scale){

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        int size = sensor.cols * sensor.rows;
        
        for (int idx = index; idx < size; idx += stride){
            int v = static_cast<int>(idx / sensor.cols);
            int u = static_cast<int>(idx - sensor.cols * v);

            int u_offset = 0;
            int v_offset = 0;
            if(scale == 2){
                u_offset = 160;
                v_offset = 120;
            }else if(scale == 4){
                u_offset = 240;
                v_offset = 180;
            }else if(scale == 8){
                u_offset = 280;
                v_offset = 210;
            }else{
                u_offset = 0;
                v_offset = 0;
            }

            if( v >= 0 && v < sensor.rows && 
                u >= 0 && u < sensor.cols ) {
                if( v % scale != 0 && u % scale != 0){
                    uint py_u = __float2uint_rd(u / scale) + u_offset;
                    uint py_v = __float2uint_rd(v / scale) + v_offset;
                    dst[py_v * sensor.cols + py_u] = src[v * sensor.cols + u];
                }
            }
        }
    }
    void pyramid::downPyramid(float *src, int scale){
        if(scale != 1){
            hipMallocManaged(&dstBlur, sizeof(float) * sensor.rows * sensor.cols);
            hipDeviceSynchronize();

            float gaussKernel[25] = {1.0,4.0,6.0,4.0,1.0,
                                    4.0,16.0,24.0,16.0,4.0,
                                    6.0,24.0,36.0,24.0,6.0,
                                    4.0,16.0,24.0,16.0,4.0,
                                    1.0,4.0,6.0,4.0,1.0};
            // Convolving guassian kenrnel on image
            convolve(src, dstBlur, gaussKernel);    // applying gaussian blur
            // removing even columns and rows in parallel from the source image
            int threads_per_block = 64;
            int thread_blocks =(sensor.cols * sensor.rows + 
                threads_per_block - 1) / threads_per_block;
            // std::cout << "<<<kernel_downSampleKernel>>> threadBlocks: "<< thread_blocks << ", threadPerBlock: " << threads_per_block << std::endl;
            downSampleKernel<<<thread_blocks, threads_per_block>>>(src, depth, sensor, scale);
            hipDeviceSynchronize();
            if(hipGetLastError())std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
            
            // hipDeviceSynchronize();
            // hipFree(dstBlur);
        }else{
            depth = src;
        }
    }
    __global__   // todo ...
    void replicationPaddingKernel(){
    }

}  // namespace DynaMap



// for (int idx = 0; idx < 100; idx++){
//     int v = static_cast<int>(idx / 10);
//     int u = static_cast<int>(idx - 10 * v);
    
//     if( v >= 0 && v < 10 && 
//         u >= 0 && u < 10 ) {
//             std:: cout << "heree .." << std::endl;
//         if( v % 2 != 0 && u % 2 != 0){
//             uint py_u = floor(u / 2);
//             uint py_v = floor(v / 2);
//             std:: cout << "pyidx: "<< py_u * 5 + py_v  << ", origidx: " << u * 10 + v<< std::endl;
//         }
//     }
// }
